#include "hip/hip_runtime.h"
//
// Created by root on 8/2/22.
//
#include <iostream>
#include <memory>
#include "cuda_common.cuh"
#include "utils.h"


const uint32_t WIDTH = 8;
const uint32_t BLOCK_SIZE = 16;
const uint32_t FULL_MASK = 0xffffffff;

void __global__ test_warp_primitives(void);

int main(int argc, char **argv)
{
    uint64_t sum = 0;
    Timing("%", for (uint64_t i = 0; i < 100000000; i ++)  sum += sum % 8 );
    Timing("&", for (uint64_t i = 0; i < 100000000; i ++)  sum += sum & 7 ); // faster
    test_warp_primitives<<<1, BLOCK_SIZE>>>();
    checkCUDA(hipGetLastError());
    checkCUDA(hipDeviceSynchronize());
    return 0;
}

void __global__ test_warp_primitives(void)
{
    int tid = threadIdx.x;
    int lane_id = tid % WIDTH;

    if (tid == 0) printf("threadIdx.x: ");
    printf("%2d ", tid);
    if (tid == 0) printf("\n");

    if (tid == 0) printf("lane_id:     ");
    printf("%2d ", lane_id);
    if (tid == 0) printf("\n");

    unsigned mask1 = __ballot_sync(FULL_MASK, tid > 0);
    unsigned mask2 = __ballot_sync(FULL_MASK, tid == 0);
    if (tid == 0) printf("FULL_MASK = %x\n", FULL_MASK);
    if (tid == 1) printf("mask1     = %x\n", mask1);
    if (tid == 0) printf("mask2     = %x\n", mask2);

    int result = __all_sync(FULL_MASK, tid);
    if (tid == 0) printf("all_sync (FULL_MASK): %d\n", result);

    result = __all_sync(mask1, tid);
    if (tid == 1) printf("all_sync     (mask1): %d\n", result); // 不能用tid = 0，因为他被屏蔽了

    result = __any_sync(FULL_MASK, tid);
    if (tid == 0) printf("any_sync (FULL_MASK): %d\n", result);

    result = __any_sync(mask2, tid);
    if (tid == 0) printf("any_sync     (mask2): %d\n", result); // 只能用tid = 0,因为只有其参数

    int value = __shfl_sync(FULL_MASK, tid, 2, WIDTH);
    if (tid == 0) printf("shfl:      ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_up_sync(FULL_MASK, tid, 1, WIDTH);
    if (tid == 0) printf("shfl_up:   ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_down_sync(FULL_MASK, tid, 1, WIDTH);
    if (tid == 0) printf("shfl_down: ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_xor_sync(FULL_MASK, tid, 1, WIDTH);
    if (tid == 0) printf("shfl_xor:  ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_up_sync(FULL_MASK, tid, 6, WIDTH * 2);
    if (tid == 0) printf("shfl_up:   ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_down_sync(FULL_MASK, tid, 6, WIDTH * 2);
    if (tid == 0) printf("shfl_up:   ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");


}
